#include "femKern.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <iostream>

int main(int argc, char const *argv[])
{
    // Define data with only 2 elements
    int numElem = 2;
    int numNodes = 64;
    int numPts = numNodes * numElem;
    int* connec = new int[numElem * numNodes];
    for (int iElem = 0; iElem < numElem; iElem++)
    {
        for (int iNode = 0; iNode < numNodes; iNode++)
        {
            connec[iElem * numNodes + iNode] = iElem * numNodes + iNode;
        }
    }
    float* f = new float[3*numPts];
    std::cout << "f = " << std::endl;
    for (int iPt = 0; iPt < numPts; iPt++)
    {
        std::cout << "[ ";
        for (int iDim = 0; iDim < 3; iDim++)
        {
            f[iDim * numPts + iPt] = 10.0f*(iDim + iPt);
            std::cout << f[iDim * numPts + iPt] << " ";
        }
        std::cout << "]" << std::endl;
    }

    // Device data
    int* d_connec;
    float* d_f;
    float* d_R;
    hipMalloc((void**)&d_connec, numElem * numNodes * sizeof(int));
    hipMalloc((void**)&d_f, 3 * numPts * sizeof(float));
    hipMalloc((void**)&d_R, numPts * sizeof(float));
    hipMemcpy(d_connec, connec, numElem * numNodes * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_f, f, 3 * numPts * sizeof(float), hipMemcpyHostToDevice);
    hipMemset(d_R, 0, numPts * sizeof(float));

    // Call the convective kernel
    fem_kern::convective<<<numElem,numNodes>>>(numPts, d_connec, d_f, d_R);
    return 0;
}