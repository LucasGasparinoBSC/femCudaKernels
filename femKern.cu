#include "hip/hip_runtime.h"
#include "femKern.cuh"

namespace fem_kern
{
    __global__ void convective(int nPt, int* connec, float* f, float* R)
    {
        int iElem = blockIdx.x; // Each block is an element
        int iPt = threadIdx.x;  // Each thread is a point or Gauss point

        // Zero the residual R
        R[connec[iElem*64 + iPt]] = 0.0f;

        // Local shared memory
        __shared__ int connec_s[64]; // Local connectivity table
        __shared__ float f_s[192];   // 3D  local flux iPt + iDim * 64
        __shared__ float div_s[64];  // Local divergence

        // Extract the global index from the connectivity table
        connec_s[iPt] = connec[iElem * 64 + iPt];
        __syncthreads();

        // Fill local flux with gobal values
        for (int iDim = 0; iDim < 3; iDim++)
        {
            f_s[iPt + iDim * 64] = f[connec_s[iPt] + iDim * nPt];
        }
        __syncthreads();

        // Compute divergence at an element
        div_s[iPt] = 0.0f;
        for (int iDim = 0; iDim < 3; iDim++)
        {
            div_s[iPt] += f_s[iPt + iDim * 64]; //! Miissinng operators, to be added later from paramms
        }
        __syncthreads();

        // Add divergence to the residual
        atomicAdd(&R[connec_s[iPt]], div_s[iPt]);
    }
}